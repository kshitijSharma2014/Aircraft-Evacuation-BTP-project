#include "hip/hip_runtime.h"
  #include<stdio.h>
  #include<stdlib.h>
  #include<time.h>
  #include"Passenger.h"
  #include"Aircraft.h"

  //this structure defines a block (it can be an exit block or a aisle block thats all
  //it also stores the id of the Passenger residing on that block)


  typedef struct block
  {
  	//Now more occ as it is useless
  int passid;	// -1 means unoccupied or Pasengerid
  int exit; // if this one is exit one
  	}block;



  // Delta z value is 1 inch so each row has 30 inch of aile length in front of it
  //

  void pass_input(Passenger P[],int n);

  __device__ void get_Aisle_Range(int range[],int i, int N)
  {

    if(i>=1 && i<=N/2 - 1)
    {
      range[0] = (i-1)*30;
      range[1] = range[0] +30 - 1;
    }
    else if(i == N/2)
    {
      range[0] = (i-1)*30;
      range[1] = range[0] + 50 -1;
    }
    else if(i == N/2 + 1)
    {
     range[0] = (i-2)*30 + 50;
     range[1] = range[0] + 50 -1;
    }
    else if(i>N/2+1 && i<=N-1)
    {
      range[0] = (i-1)*30+40;
      range[1] = range[0] +30 - 1;
    }
  //return range;
  }





/*
  __device__ int select_exit(Passenger P, int exit[])
  {
      int ans =0;
      if ((exit[0] == 1 || exit[1] == 1) && (exit[2]==1 || exit[3]==1 || exit[4]==1 || exit[5]==1) && P.x<=470){
          if(P.x<470-P.x)
          {
              ans = (exit[0] == 1) ? 0:1;
          }
          else{
              if(exit[2]==1 || exit[3]==1){
                  ans = (exit[2] == 1) ? 2:3;
              }
              else if(exit[4]==1 || exit[5]==1){
                  ans = (exit[4] == 1) ? 4:5;
              }
          }
      }
      else if ((exit[6] == 1 || exit[7] == 1) && (exit[2]==1 || exit[3]==1 || exit[4]==1 || exit[5]==1) && P.x>470){
          if(940-P.x<P.x-470){
              ans = (exit[6] == 1) ? 6:7;
          }
          else{
              if(exit[4]==1 || exit[5]==1){
                  ans = (exit[4] == 1) ? 4:5;
              }
              else if(exit[2]==1 || exit[3]==1){
                  ans = (exit[2] == 1) ? 2:3;
              }
          }

      }
      return ans;
  }

*/
__device__ int select_exit(Passenger P, int exit[])
  {
      int i,ans =-1;
      if(P.x<470)
      {
        if(P.x<470-P.x)
        {
          if(exit[0]==1||exit[1]==1)
            ans=0;
        }
        else
        {
          if(exit[2]==1||exit[3]==1||exit[4]==1||exit[5]==1)
            ans=3;
        }
      }
      else
      {
        if(940-P.x<P.x-470)
        {
          if(exit[7]==1||exit[8]==1)
            ans=7;
        }
        else
        {
          if(exit[2]==1||exit[3]==1||exit[4]==1||exit[5]==1)
            ans=3;
        }
      }
      if(ans==-1)
      {
        for(i=0;i<6;++i)
          {
            if(exit[i]==1)
              return i;
          }
      }
      else
        return ans;
  }


  __device__ int get_direction(Passenger p, block A[], int exitnum){
      if(exitnum == 0 || exitnum == 1){
          if(p.x-4 > 0){
              return 1;
          }
          else if(p.x-4 <= 0){
              return 0;
          }
      }
      else if (exitnum == 6 || exitnum == 7)
      {
          if(p.y+4 < 940){
              return -1;
          }
          else if(p.y >= 940){
              return 0;
          }
      }
      else if ((exitnum == 2 || exitnum == 3))
      {
          if(p.x < 450){
              return -1;
          }
          if(p.x > 500){
              return 1;
          }
          if(p.x >= 450 || p.x <= 500){
              return 0;
          }
      }
      else if ((exitnum == 4 || exitnum == 5))
      {
          if(p.x < 500){
              return -1;
          }
          if(p.x > 550){
              return 1;
          }
          if(p.x >= 500 || p.x <= 550){
              return 0;
          }
      }
      return 0;
  }
 /*
  __global__ void map_Passenger_to_exit(Passenger P[], int seat[100][100], block C[][55],int exit[]){

    int k,j,m,i =2, rownm;

    while(i<6 && exit[i]!=0){
      if(i==2)
      {
          rownm = 15;
          j=0;
      }
      if(i==3)
      {
          rownm = 15;
          j=5;
      }
      if(i==4)
      {
          rownm = 16;
          j=0;
      }

      if(i==5){
          rownm = 16;
          j=5;
      }
      for(k=0;k<35;){

          for( m = k; m < k + (int)P[ seat[rownm][j] ].diameter ; ++m){
              C[i-2][m].passid = seat[rownm][j];
          }
          seat[rownm][j] = 0;
          //C[i-2][p[seat[rownm][j]].diameter-1].passid = -1*seat[rownm][j];
          if(i==2 || i==4){
              ++j;
          }
          else if(i==3 || i==5){
              --j;
          }
          k=k+17;
      }
      ++i;
    }
  }
*/




  __global__ void movement_to_exit(block A[],block B[4][55],block C[4][55],Passenger P[] ,int seat[][100],int d_exit[],int numPass) //runs for each Passenger and make his movmenent according to the positions
  {
  // Now we have to map the thread id with the passennger id

  int i=threadIdx.x,k;
  int range[2];
  int j,count=0;
  int tex,ex,dir;
//  int exit[]={1,1,0,0,1,1,0,0};

  if(i < numPass)
  {
   switch(P[i].status)
   {
    //printf("Hello\n" );
   	case 0: //the Passenger is in his seat aisle (x=row  number y = (1-6)column in seat )
      if(P[i].y == 2 || P[i].y==3)
   		{

  			get_Aisle_Range(range, P[i].x , 30);
  		//	range[0]=0;
       // range[1]=50;
        count=0;
  			for(j=range[0];j<range[1];++j)
  				{
  					if(A[j].passid == -1)
  					{
  						count++;
  						if(count >= P[i].diameter)
  							break;
  					}
  					else
  						count=0;
  				}

  				if(j<range[1]+1)
  				{

  					for(k=j;k>=j-P[i].diameter;--k)
  						A[k].passid=i;
  					seat[P[i].x][P[i].y]=-1;
            P[i].x=k;
  					P[i].y=j;
  					P[i].status=1;
            P[i].res=0;
  				}
      }
   		else
   			{
   				if(P[i].y < 2)
   				{
   					if(seat[P[i].x][P[i].y+1]==-1)
   						{
   				     if(P[i].res==60)
          			{
                  P[i].y++;
   							seat[P[i].x][P[i].y-1]=-1;
   							seat[P[i].x][P[i].y] = i;//
   						   P[i].res=0;
                }
                else
                  P[i].res++;
            }

   				}
   				else
   				{
   					if(seat[P[i].x][P[i].y-1]==-1)
   						{
                if(P[i].res==60)
                {
   							  P[i].y--;
   							  seat[P[i].x][P[i].y+1]=-1;
   							  seat[P[i].x][P[i].y] = i;//
   						   P[i].res=0;
                }
                else
                  P[i].res++;
              }

   				}
   			}
   	break;
  //done till here ;D


  //comment starts here:
   	case 1:
   		//	the Passenger is in aisle and here the  x value that tell the starting of the Passenger
   		//  y   is the ending point of the Passenger
   		//	Select the exit and try to move towards the aisle point of that exit

      ex = select_exit(P[i], d_exit); // Create an exit array that contain 0 if the exit is not open and 1 if it is open
   		dir= get_direction(P[i],A,ex);

      P[i].ans=ex;
      P[i].dir=dir;

      if(dir == 1)
   		{

   			//move up
   			if(A[P[i].x-2].passid == -1)
   			{
   				if(P[i].speed==1.0f)
   				{
            if(P[i].res==2)
            {
   					  P[i].x-=2;
   					  P[i].y-=2;

   					  A[P[i].x].passid  = i;
   					  A[P[i].x+1].passid  = i;

   					  A[P[i].y+1].passid  = -1;
   					  A[P[i].y+2].passid  = -1;
   				   P[i].res=0;
           }
           else
            P[i].res++;
          }
   				else
   				{
   					if(P[i].speed==1.5f && A[P[i].x-3].passid == -1)
   					{
              if(P[i].res==2)
              {
     						 P[i].x-=3;
     						 P[i].y-=3;

     						A[P[i].x+1].passid  = i;
     						A[P[i].x+2].passid  = i;
     						A[P[i].x].passid  = i;

     						A[P[i].y+1].passid  = -1;
     						A[P[i].y+2].passid  = -1;
     						A[P[i].y+3].passid  = -1;
                P[i].res=0;
              }
              else
                P[i].res++;
   					  }
              else
                P[i].res=0;

   				}
    			}
          else
              P[i].res=0;
    			//else dont move
   		}
   		else
   		{
   			if(dir==-1)
   			{
   				//move down
   				if(A[P[i].y+2].passid == -1)
   				{
   					if(P[i].speed==1.0f)
   					{
              if(P[i].res==2)
              {
   						P[i].x+=2;
   						P[i].y+=2;

   						A[P[i].y].passid  = i;
   						A[P[i].y-1].passid  = i;

   						A[P[i].x-1].passid  = -1;
   						A[P[i].x-2].passid  = -1;
              P[i].res=0;
   					  }
              else
                P[i].res++;
            }
   					else
   					{
   						if(A[P[i].y+3].passid == -1)
   						{
                if(P[i].res==2)
                {
   							  P[i].x+=3;
     							P[i].y+=3;

     							A[P[i].y].passid  = i;
     							A[P[i].y-2].passid  = i;
     							A[P[i].y-1].passid  = i;

     							A[P[i].x-1].passid  = -1;
     							A[P[i].x-2].passid  = -1;
     							A[P[i].x-3].passid  = -1;
                  P[i].res=0;
     						}
                else
                  P[i].res++;
                }
              else
                P[i].res=0;

   				}
    				}
            else
              P[i].res=0;

   			}
   			else
   			{
          P[i].res=0;
   				//stay and jump to B or C
   				if(ex==0||ex==1||ex==6||ex==7)
   				{
   					//Going to B
            tex=ex;
   					if(ex==6||ex==7)
   						tex=ex-5;

            // Going to B[tex]

   					for(j=50;B[tex][j].passid==-1&&j> 50-P[i].diameter ;--j);

   					if(50 - j == P[i].diameter)
   					{
   						for(k=P[i].x;k<=P[i].y;++k)
                A[k].passid=-1;
              P[i].x = tex;
   						P[i].y = j;

   					for(;j<=50;++j)
   						B[tex][j].passid=i;
            P[i].status=2;
   				 }
          }
   				else
   				{
            P[i].res=0;
              tex=ex-2;

            // Going to B[tex]

            for(j=50;C[tex][j].passid==-1&&j>50-P[i].diameter;--j);

            if(50 - j == P[i].diameter)
            {
              for(k=P[i].x;k<=P[i].y;++k)
                A[k].passid=-1;
              P[i].x = tex;
              P[i].y = j;

            for(;j<=50;++j)
              C[tex][j].passid=i;
            P[i].status=3;
          }
   				}
   			}

   		}
   	break;


    case 2: // the Passenger is in midle of the exit front and end exit aisles i.e seat exit y represent the position in the aisle and
   		// x represent which aisle 1 / 2 / 3 / 4
   		if(P[i].y <= 0||(P[i].speed==1.0f && P[i].y-2 <= 0)||(P[i].speed==1.5f && P[i].y-3 <= 0) )
   		{
   	    for(j=0;j<10; ++j)
          {
            if(B[P[i].x][j].passid ==i)
              B[P[i].x][j].passid=-1;
        }
        P[i].status = 4; //Passenger is out of the plane
    	}
   		else
   		{

   			if(P[i].speed == 1.0f)
   			{
  				if(B[P[i].x][P[i].y-2].passid ==-1)
  				{
  	 				//move closer to the exit

  					P[i].y-=2;

  	 				B[P[i].x][P[i].y].passid = i;
  	 				B[P[i].x][P[i].y + 1].passid = i;

  	 				B[P[i].x][(P[i].y + (int)P[i].diameter + 1)].passid =-1;
  	 				B[P[i].x][(P[i].y + (int)P[i].diameter + 2)].passid =-1;

  	 			}
  	 		}
  	 		else
  	 		{
  	 			if(B[P[i].x][P[i].y -3 ].passid ==-1)
  	 			{
  	 				//move closer to the exit
  	 				P[i].y-=3;

  	 				B[P[i].x][P[i].y].passid = i;
  	 				B[P[i].x][P[i].y + 2].passid = i;
  	 				B[P[i].x][P[i].y + 1].passid = i;

  	 				B[P[i].x][P[i].y + (int)P[i].diameter + 1].passid =-1;
  	 				B[P[i].x][P[i].y + (int)P[i].diameter + 2].passid =-1;
  	 				B[P[i].x][P[i].y + (int)P[i].diameter + 3].passid =-1;


  	 			}
  	 		}
   		}

   	break;
   	case 3: // the Passenger is in midle of the midle exit aisles i.e seat exit y represent the position in the aisle and
   		// x represent which aisle 1 / 2 / 3 / 4
   	if(P[i].y <= 0||(P[i].speed==1.0f && P[i].y-2 <= 0)||(P[i].speed==1.5f && P[i].y-3 <= 0) )
   		{
   			for(j=0;j<10; ++j)
   				{
            if(C[P[i].x][j].passid ==i)
              C[P[i].x][j].passid=-1;
   			}
        P[i].status = 4; //Passenger is out of the plane
   		}
   		else
   		{

   			if(P[i].speed == 1.0f)
   			{
  				if(C[P[i].x][P[i].y-2].passid ==-1)
  				{
  	 				//move closer to the exit
  	 			 if(P[i].res==1)
           {
          	P[i].y-=2;
  	 				C[P[i].x][P[i].y ].passid = i;
  	 				C[P[i].x][P[i].y + 1].passid = i;


  	 				C[P[i].x][P[i].y + (int)P[i].diameter + 1].passid =-1;
  	 				C[P[i].x][P[i].y + (int)P[i].diameter + 2].passid =-1;
          }
          else
            P[i].res++;
  	 			}
          else
            P[i].res=0;
  	 		}
  	 		else
  	 		{
  	 			if(P[i].speed == 1.5f&&C[P[i].x][P[i].y -3].passid ==-1)
  	 			{
  	 				//move closer to the exit
            if(P[i].res==1)
            {
  	 				P[i].y-=3;

  	 				C[P[i].x][P[i].y ].passid = i;
  	 				C[P[i].x][P[i].y + 2].passid = i;
  	 				C[P[i].x][P[i].y + 1].passid = i;


  					C[P[i].x][P[i].y + (int)P[i].diameter + 1].passid =-1;
  	 				C[P[i].x][P[i].y + (int)P[i].diameter + 2].passid =-1;
  	 				C[P[i].x][P[i].y + (int)P[i].diameter + 3].passid =-1;
            }
            else
              P[i].res++;
  	 			}
          else
            P[i].res=0;
  	 		}
   		}

   	break;
    case 4:
      P[i].x=-1;
      P[i].y=-1;
   };

  }

  }





void map_Passenger_to_exit(Passenger P[],int seat[][100], block C[][55],block B[][55],int h_exit[])
{
  int i,k,j,l;
  for(l=2;l<6;++l)
  {
    if(h_exit[l]==1)//the middle exit 2,3,4,5
    {
      if(l%2==0)
        {
          for(i=0;i<3;i++)
          {
            k=seat[15+(l-2)/2][i];
            if(k!=-1)
              {
                for(j=0;j<P[k].diameter;++j)
                  C[l-2][i*17 + j].passid=k;

                P[k].x=l-2;
                P[k].status = 3;
                P[k].y= i*17;
                P[k].res=0;
              }
            }
        }
        else
        {
          for(i=3;i<6;i++)
          {
            k=seat[15+(l-2)/2][i];
            if(k!=-1)
              {
                for(j=0;j<P[k].diameter;++j)
                  C[l-2][(5-i)*17 + j].passid=k;

                P[k].x=l-2;
                P[k].status = 3;
                P[k].y= (5-i)*17;
                P[k].res=0;
              }
            }
        }

    }
  }

}




  //Main
  int main()
  {
    srand(time((0)));
    //Aircraft* air= input();
    int numPass,i,j;
    int count1=0;
    scanf("%d",&numPass);
    Passenger *h_P =(Passenger *)malloc(sizeof(Passenger)*numPass);
    Passenger *P;


    pass_input(h_P,numPass);


    char aircraftName[30];
    scanf("%s",aircraftName);
    Aircraft A;
    aircraftInput(A,aircraftName);

  	// Seating Arrangement Assigning Each Passenger location to sit randomely
  	// Think something to make sure the random function does not send it to infinite loop

  	int h_seat[A->row][A->column];  // initialise exact array (compile with check bound)  check bound
  	int (*seat)[100];

  	for(i=0;i<A->row;++i)
  	{
  		for(j=0;j<A->column;++j)
  			h_seat[i][j]=-1;
  	}

  	//all seats are vacant right now
//TODO start id with 1
  	int r_row,r_col;
  	for(i=0;i<numPass;++i)
  	{
  		r_row=rand()%30;   //should be defined in the header file TODO
  		r_col=rand()%6;     //should be defined in the header file TODO
  		if(h_seat[r_row][r_col]==-1)
  			{
  					h_P[i].x=r_row;
  					h_P[i].y=r_col;
  			    if(r_row<0||r_col<0)
              {
                printf("Olala\n");
                return 0;
              }
            else
              h_seat[r_row][r_col]=i;
            printf("%d %d\n",r_row,r_col);
        }
  		else
  		{
  				i--;
  			}


  	}
   // for(i=0;i<30;++i)
    //{
     // for(j=0;j<10;++j)
      //  printf("%d ",h_seat[i][j]+1);
      //printf("\n");
   // }



  	// Now each row is occupied by some Passengers
  	// Each Passenger is sitting in a row and each row is having corresponding aisle array portion in front of it.
  	// The Passenger can move to the aisle A[] in front of its row if it is unoccupied
  printf("Seating Done\n");
int aisleLength = ((A->row - 2) * 30 ) + 100);
  	block h_A[aisleLength]; //length of aisle should be general
  	block* A;


  	// A is the aisle
  	//Each Element of the
  	for(j=0;j<aisleLength;++j)
  		{
  				h_A[j].passid=-1;
  				h_A[j].exit=0;
  		}
  	// the aisle is empty right now
  	//Now there are 4 Normal Gate Exits and
  	//Exit Paths are of 2 types 1 end and other in the middle each one will have different speeds


  	block h_B[4][55]; // Nornal Exit Paths 2 on each ends of the plane
  	block h_C[4][55]; // Seat Exit 2 in the middle of the plane

  	block (*B)[55]; // Nornal Exit Paths 2 on each ends of the plane
  	block (*C)[55]; // Seat Exit 2 in the middle of the plane

  	for(i=0;i<4;++i)
  	{
  		for(j=0;j<55;++j)
  			{
  				h_B[i][j].passid=-1;
  				h_C[i][j].passid=-1;
  				h_B[i][j].exit=0;
  				h_C[i][j].exit=0;
  			}
  	}

  	// set up the exits all the B exits are empty
  	// C exits or the middle exits are occupied by people
  	//int h_exit[6] = {1,1,1,1,1,1};
  	//exit is 1 for those wxits which are open and 0 for those which are close
  	//block A[],block B[4][55],block C[4][55],Passenger P[] ,int seat[][100],int numPass
  	//hipMalloc((void **) &array1_d , WIDTH*WIDTH*sizeof (int) ) ;
  	// Here the game starts
  	//Emergency! Emergency! Emergency! Run all of you Out of the plane
    printf("Enter 1 if the exit is open and 0 if the exit is close for all the 8 exits");
    int h_exit[8];
    int *d_exit;
    for(i=0;i<8;++i)
      scanf("%d",&h_exit[i]);

  	//__global__ void movement_to_exit(block A[],block B[4][55],block C[4][55],Passenger P[] ,int seat[][100],int numPass) //runs for each Passenger and make his movmenent according to the positions
  	int numout=0,numprev=0;
  	j=0;  //change the variable name to some specific time var
    hipMalloc((void **) &P , numPass*sizeof (Passenger) ) ;
    hipMalloc((void **) &B , (55*4)*sizeof (block) ) ;
    hipMalloc((void **) &C , (55*4)*sizeof (block) ) ;
    hipMalloc((void **) &A , 1000*sizeof (block) ) ;
    hipMalloc((void **) &seat , (100*100)*sizeof (int) ) ;
    hipMalloc((void **) &d_exit , (8)*sizeof (int) ) ;

    // select exit
    map_Passenger_to_exit(h_P,h_seat, h_C,h_B,h_exit);
    for(j=0;j<4;++j)
    {
      for(i=0;i<55;++i)
      {
       printf("%d",h_C[j][i]);
      }
      printf("\n");
    }
    count1=0;
    int filecounter = 0;
  	while(numout<numPass)
  	{ ++filecounter;
     // if(j==1000)
      //  break;
      numprev=numout;
  		numout=0;

   /* for(i=0;i<30;++i)
    {
      for(j=0;j<10;++j)
        printf("%d ",h_seat[i][j]+1);
      printf("\n");
    }
    */

/*    for(i=0;i<numPass;++i)
      {
        if(P[i].status!=4)
        printf("Passengr %d : (%d,%d) : %d : ans : %d : dir : %d\n",i,h_P[i].x,h_P[i].y,h_P[i].status,h_P[i].ans,h_P[i].dir);
      }

  */
    //printf("Passengr %d : (%d,%d) : %d : ans : %d : dir : %d\n",3,h_P[3].x,h_P[3].y,h_P[3].status,h_P[3].ans,h_P[3].dir);

    hipMemcpy ( P , h_P , numPass*sizeof (Passenger) , hipMemcpyHostToDevice);
    hipMemcpy ( seat , h_seat , 100*100*sizeof (int) , hipMemcpyHostToDevice);
    hipMemcpy ( C , h_C , 4*55*sizeof (block) , hipMemcpyHostToDevice);
    hipMemcpy ( B , h_B , 4*55*sizeof (block) , hipMemcpyHostToDevice);
    hipMemcpy ( A , h_A , 1000*sizeof (block) , hipMemcpyHostToDevice);
    hipMemcpy ( d_exit , h_exit , 8*sizeof (int) , hipMemcpyHostToDevice);


      movement_to_exit<<< 1,numPass >>>(A,B,C,P,seat,d_exit,numPass);

  		hipError_t err1 = hipPeekAtLastError();
      hipDeviceSynchronize();
      //printf( "Got CUDA error ... %s \n", hipGetErrorString(err1));

    hipMemcpy ( h_P , P , numPass*sizeof (Passenger) , hipMemcpyDeviceToHost);
    hipMemcpy ( h_seat , seat , 100*100*sizeof (int) , hipMemcpyDeviceToHost);
    hipMemcpy ( h_C , C , 4*55*sizeof (block) , hipMemcpyDeviceToHost);
    hipMemcpy ( h_B , B , 4*55*sizeof (block) , hipMemcpyDeviceToHost);
    hipMemcpy ( h_A , A , 1000*sizeof (block) , hipMemcpyDeviceToHost);
// creating file
   FILE *fp;

    char filename[] = "output";
    char str[100];
    sprintf(str, "%d", filecounter);
    strcat(filename,str);
    strcat(filename,".txt");


    fp = fopen(filename, "w");
    int global[1000][150];
    for (int i = 0; i <1000; ++i)
    {
      for (int j = 0; j < 150; ++j)
      {
        fprintf(fp, "%d", global[i][j]);
      }
      fprintf(fp, "\n", );
    }

    fprintf(fp, "This is testing...\n");



    fclose(fp);

//file creation complete


  		for(i=0;i<numPass;i++)
  		{
  			if(h_P[i].status == 4)
  				numout++;
  	//	  printf("%d\n",h_P[i].status);
      }

    if(numprev==numout)
      {
        printf("*");
        count1++;
      }
    else
    {
      count1=0;
      printf("%d %d\n",numout,j);
    }
      if(numout==numPass)
        break;

    //printf("%d %d\n",numout,j);

      if(count1>100)
      {
        for(i=0;i<numPass;i++)
        {
          if(h_P[i].status!=4)
          {
                printf("Passengr %d : (%d,%d) : %d : ans : %d : dir : %d\n",i,h_P[i].x,h_P[i].y,h_P[i].status,h_P[i].ans,h_P[i].dir);
          }
        }
      //  break;
      }

     // printf("%d\t %d\n",numout,j);
  		j++;
  	}

  	float timeSteps = 40.6;
  	printf("%f\n",j*timeSteps + 7000.0);
  //  printf("%f\n",j*timeSteps);
  	return 0;
  }


  int random(int min,int max);

  void pass_input(Passenger *P,int n)
  {
    int i;//r;
    srand(time(0));
    Passenger *tp=P;

    for(i=0;i<n;++i,tp++)
      {
        //r=rand();
        //  printf("%d\n",r);
        tp->id=i;
        //  tp->x=
        //
        tp->sex=random(0,1); // Male or female(random 0-1)
        tp->status = 0;
        tp->Mtime=tp->sex?random(875,1750):random(920,1950);
        tp->Wtime=50;
        tp->Rtime=random(400,700); // Random  (500-1000)ms
        tp->fear=-1; //fear value 0
        tp->agility=-1; // agility value

        tp->diameter=random(9,15);//(Random ) // diameter occupied by passenger
        tp->totaltime=0; //total time to evacuate
        tp->totalDist=0; //total distance to exit
        if(tp->sex==0){
          float t = (float)random(10,15);
         tp->speed=t/10;  //Random (1-1.5 ) speed of passenger
        }
        else{
          float t = (float)random(9,12);
          tp->speed=t/10;
        }
        tp->grpstatus=-1; // Not in this paper in group or not
        tp->timeSteps=178; // minimum unit of time = 178 miliseconds
        tp->res=0;
        printf("id : %d , sex : %d , Mtime : %d, Rtime : %d\n",tp->id,tp->sex,tp->Mtime,tp->Rtime);
      }

  }

  Aircraft aircraftInput(Aircraft *A,char name[]){
    char filename1[100];
    strcat(filename1,".txt");


   FILE *fp;
   char buff[255];
   int res[10];
   int k =0;
   fp = fopen(filename1, "r");
   while(fgets(buff, 80, fp) != NULL)
   {
   //fscanf(fp, "%s", buff);
   //printf("%s\n", buff );
   int result = atoi(buff);
   //printf("%d\n", result);
    res[k] = result;
    ++k;

}
    fclose(fp);
    A->row = res[0];
    A->column = res[1];
    A->numOfExitPassage = res[2];
    A->maxNumPassenger = res[3];
}

void createGlobalMatrix(int global[1000][150], block h_seat, block h_A, block h_B, block h_C){

    //main exit 1
    int k =0, i=0;
    for(k =0;k<50;++k){
      for (i = 0; i < 55; ++i)
      {
          if(h_B[0][i].passid == -1){
            global[k][i] = 0;
          }
          else{
            global[k][i] = 1;
          }
        }
        int temp = i;
        if(h_A[k].passid == -1){
            global[k][i] = 0;
          }
          else{
            global[k][i] = 1;
          }

      for (i = 54; i >=0; --i)
      {
          if(h_B[1][i].passid == -1){
            global[k][i] = 0;
          }
          else{
            global[k][i] = 1;
          }
        }
      }

      // 1-14 seats
      int p = 0;
      for (i = 50; i < (14*30)+50; ++i)
      {
          int u=0;
        for (int j = 0; j < 3; ++j)
        {
          if (h_seat[p][j] == -1)
          {
            for (; u < 18*(j+1); ++u)
            {
              global[i][u] = 0;
            }
          }
          else{
              for (; u < 18*(j+1); ++u)
              {
                global[i][u] = 1;
              }
          }
        }
        global[i][u] = h_A[i];

        int u=0;
        for (int j = 3; j < 6; ++j)
        {

          if (h_seat[p][j] == -1)
          {
            for (; u < 18*(j+1); ++u)
            {
              global[i][u] = 0;
            }
          }
          else{
              for (; u < 18*(j+1); ++u)
              {
                global[i][u] = 1;
              }
          }
        }
        if ((i+1-50)%30 == 0)
        {
          ++p;
        }
      }


      //middle exits

      for(k =i;k<i+50;++k){
      int i =0;
      for (j = 0; j < 55; ++j)
      {
          if(h_C[0][j].passid == -1){
            global[k][j] = 0;
          }
          else{
            global[k][j] = 1;
          }
        }
        int temp = i;
        if(h_A[k].passid == -1){
            global[k][j] = 0;
          }
          else{
            global[k][j] = 1;
          }

      for (j = 54; j >=0; --j)
      {
          if(h_C[1][j].passid == -1){
            global[k][j+1] = 0;
          }
          else{
            global[k][j+1] = 1;
          }
        }
      }

      // middle exits  correct the second loop for j.. j is not in the sync

      for(;k<i+100;++k){
      int i =0;
      for (j = 0; j < 55; ++j)
      {
          if(h_C[2][j].passid == -1){
            global[k][j] = 0;
          }
          else{
            global[k][j] = 1;
          }
        }
        int temp = i;
        if(h_A[k].passid == -1){
            global[k][j] = 0;
          }
          else{
            global[k][j] = 1;
          }

      for (j = 54; j >=0; --j)
      {
          if(h_C[3][j].passid == -1){
            global[k][j+1] = 0;
          }
          else{
            global[k][j+1] = 1;
          }
        }
      }


      // 17 - 30 seats

      for (i = k; i < (28*30)+150; ++i)
      {
          int u=0;
        for (int j = 0; j < 3; ++j)
        {
          if (h_seat[p][j] == -1)
          {
            for (; u < 18*(j+1); ++u)
            {
              global[i][u] = 0;
            }
          }
          else{
              for (; u < 18*(j+1); ++u)
              {
                global[i][u] = 1;
              }
          }
        }
        global[i][u] = h_A[i];

        int u=0;
        for (int j = 3; j < 6; ++j)
        {

          if (h_seat[p][j] == -1)
          {
            for (; u < 18*(j+1); ++u)
            {
              global[i][u] = 0;
            }
          }
          else{
              for (; u < 18*(j+1); ++u)
              {
                global[i][u] = 1;
              }
          }
        }
        if ((i+1-150)%30 == 0)
        {
          ++p;
        }
      }


//end exits
      int q = i+50;
      for(k =i;k<q;++k){
      int i =0;
      for (i = 0; i < 55; ++i)
      {
          if(h_B[2][i].passid == -1){
            global[k][i] = 0;
          }
          else{
            global[k][i] = 1;
          }
        }
        int temp = i;
        if(h_A[k].passid == -1){
            global[k][i] = 0;
          }
          else{
            global[k][i] = 1;
          }

      for (i = 54; i >=0; --i)
      {
          if(h_B[3][i].passid == -1){
            global[k][i] = 0;
          }
          else{
            global[k][i] = 1;
          }
        }
      }


}

  int random(int min,int max)
  {
    int r = rand();
    r=r%(max-min+1);
    r=r+min;
    return r;
  }
