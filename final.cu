#include "hip/hip_runtime.h"
  #include<stdio.h>
  #include<stdlib.h>
  #include<time.h>
  #include"Passenger.h"
  #include"Aircraft.h"

  //this structure defines a block (it can be an exit block or a aisle block thats all
  //it also stores the id of the Passenger residing on that block)


  typedef struct block
  {
  	//Now more occ as it is useless
  int passid;	// -1 means unoccupied or Pasengerid
  int exit; // if this one is exit one
  	}block;



  // Delta z value is 1 inch so each row has 30 inch of aile length in front of it
  //
void printToFile(char filename[], int a[]);
  void pass_input(Passenger P[],int n);
void createGlobalMatrix(int h_seat[100][100], block h_A[1000], block h_B[4][55], block h_C[4][55], char filename[50],float tt);
Aircraft aircraftInput(char name[]);
  __device__ void get_Aisle_Range(int range[],int i, int N)
  {

    if(i>=1 && i<=N/2 - 1)
    {
      range[0] = (i-1)*30;
      range[1] = range[0] +30 - 1;
    }
    else if(i == N/2)
    {
      range[0] = (i-1)*30;
      range[1] = range[0] + 50 -1;
    }
    else if(i == N/2 + 1)
    {
	range[0] = (i-2)*30 + 50;
	range[1] = range[0] + 50 -1;
    }
    else if(i>N/2+1 && i<=N-1)
    {
      range[0] = (i-1)*30+40;
      range[1] = range[0] +30 - 1;
    }
  //return range;
  }





/*
  __device__ int select_exit(Passenger P, int exit[])
  {
      int ans =0;
      if ((exit[0] == 1 || exit[1] == 1) && (exit[2]==1 || exit[3]==1 || exit[4]==1 || exit[5]==1) && P.x<=470){
          if(P.x<470-P.x)
          {
              ans = (exit[0] == 1) ? 0:1;
          }
          else{
              if(exit[2]==1 || exit[3]==1){
                  ans = (exit[2] == 1) ? 2:3;
              }
              else if(exit[4]==1 || exit[5]==1){
                  ans = (exit[4] == 1) ? 4:5;
              }
          }
      }
      else if ((exit[6] == 1 || exit[7] == 1) && (exit[2]==1 || exit[3]==1 || exit[4]==1 || exit[5]==1) && P.x>470){
          if(940-P.x<P.x-470){
              ans = (exit[6] == 1) ? 6:7;
          }
          else{
              if(exit[4]==1 || exit[5]==1){
                  ans = (exit[4] == 1) ? 4:5;
              }
              else if(exit[2]==1 || exit[3]==1){
                  ans = (exit[2] == 1) ? 2:3;
              }
          }

      }
      return ans;
  }

*/
__device__ int select_exit(Passenger P, int exit[])
  {
      int i,ans =-1;
      if(P.x<470)
      {
        if(P.x<470-P.x)
        {
          if(exit[0]==1||exit[1]==1)
            ans=0;
        }
        else
        {
          if(exit[2]==1||exit[3]==1||exit[4]==1||exit[5]==1)
            ans=3;
        }
      }
      else
      {
        if(940-P.x<P.x-470)
        {
          if(exit[7]==1||exit[8]==1)
            ans=7;
        }
        else
        {
          if(exit[2]==1||exit[3]==1||exit[4]==1||exit[5]==1)
            ans=3;
        }
      }
      if(ans==-1)
      {
        for(i=0;i<6;++i)
          {
            if(exit[i]==1)
              return i;
          }
      }
      else
        return ans;
  }


  __device__ int get_direction(Passenger p, block A[], int exitnum){
      if(exitnum == 0 || exitnum == 1){
          if(p.x-4 > 0){
              return 1;
          }
          else if(p.x-4 <= 0){
              return 0;
          }
      }
      else if (exitnum == 6 || exitnum == 7)
      {
          if(p.y+4 < 940){
              return -1;
          }
          else if(p.y >= 940){
              return 0;
          }
      }
      else if ((exitnum == 2 || exitnum == 3))
      {
          if(p.x < 450){
              return -1;
          }
          if(p.x > 500){
              return 1;
          }
          if(p.x >= 450 || p.x <= 500){
              return 0;
          }
      }
      else if ((exitnum == 4 || exitnum == 5))
      {
          if(p.x < 500){
              return -1;
          }
          if(p.x > 550){
              return 1;
          }
          if(p.x >= 500 || p.x <= 550){
              return 0;
          }
      }
      return 0;
  }
 /*
  __global__ void map_Passenger_to_exit(Passenger P[], int seat[100][100], block C[][55],int exit[]){

    int k,j,m,i =2, rownm;

    while(i<6 && exit[i]!=0){
      if(i==2)
      {
          rownm = 15;
          j=0;
      }
      if(i==3)
      {
          rownm = 15;
          j=5;
      }
      if(i==4)
      {
          rownm = 16;
          j=0;
      }

      if(i==5){
          rownm = 16;
          j=5;
      }
      for(k=0;k<35;){

          for( m = k; m < k + (int)P[ seat[rownm][j] ].diameter ; ++m){
              C[i-2][m].passid = seat[rownm][j];
          }
          seat[rownm][j] = 0;
          //C[i-2][p[seat[rownm][j]].diameter-1].passid = -1*seat[rownm][j];
          if(i==2 || i==4){
              ++j;
          }
          else if(i==3 || i==5){
              --j;
          }
          k=k+17;
      }
      ++i;
    }
  }
*/



    __global__ void movement_to_exit(block A[],block B[4][55],block C[4][55],Passenger P[] ,int seat[][100],int d_exit[],int numPass) //runs for each Passenger and make his movmenent according to the positions
    {
    // Now we have to map the thread id with the passennger id

    int i=threadIdx.x,k;
    int range[2];
    int j,count=0;
    int tex,ex,dir;
  //  int exit[]={1,1,0,0,1,1,0,0};

    if(i < numPass)
    {
     switch(P[i].status)
     {
      //printf("Hello\n" );
     	case 0: //the Passenger is in his seat aisle (x=row  number y = (1-6)column in seat )
        if(P[i].y == 2 || P[i].y==3)
     		{

    			get_Aisle_Range(range, P[i].x , 30);
    		//	range[0]=0;
         // range[1]=50;
          count=0;
    			for(j=range[0];j<range[1];++j)
    				{
    					if(A[j].passid == -1)
    					{
    						count++;
    						if(count >= P[i].diameter)
    							break;
    					}
    					else
    						count=0;
    				}

    				if(j<range[1]+1)
    				{

    					for(k=j;k>=j-P[i].diameter;--k)
    						A[k].passid=i;
    					seat[P[i].x][P[i].y]=-1;
              P[i].x=k;
    					P[i].y=j;
    					P[i].status=1;
              P[i].res=0;
    				}
        }
     		else
     			{
     				if(P[i].y < 2)
     				{
     					if(seat[P[i].x][P[i].y+1]==-1)
     						{
     				     if(P[i].res==60)
            			{
                    P[i].y++;
     							seat[P[i].x][P[i].y-1]=-1;
     							seat[P[i].x][P[i].y] = i;//
     						   P[i].res=0;
                  }
                  else
                    P[i].res++;
              }

     				}
     				else
     				{
     					if(seat[P[i].x][P[i].y-1]==-1)
     						{
                  if(P[i].res==60)
                  {
     							  P[i].y--;
     							  seat[P[i].x][P[i].y+1]=-1;
     							  seat[P[i].x][P[i].y] = i;//
     						   P[i].res=0;
                  }
                  else
                    P[i].res++;
                }

     				}
     			}
     	break;


    //comment starts here:
     	case 1:
     		//	the Passenger is in aisle and here the  x value that tell the starting of the Passenger
     		//  y   is the ending point of the Passenger
     		//	Select the exit and try to move towards the aisle point of that exit

        ex = select_exit(P[i], d_exit); // Create an exit array that contain 0 if the exit is not open and 1 if it is open
     		dir= get_direction(P[i],A,ex);

        P[i].ans=ex;
        P[i].dir=dir;

        if(dir == 1)
     		{

     			//move up
     			if(A[P[i].x-1].passid == -1 || A[P[i].x-2].passid == -1 || A[P[i].x-3].passid == -1 || A[P[i].x-4].passid == -1)
     			{
     				if(P[i].speed!=0.0f)
     				{
              if(P[i].res==2)
              {
     					  P[i].x-=2*P[i].speed;
     					  P[i].y-=2*P[i].speed;

                for (int s = 0; s < 2*P[i].speed; ++s)
                {
                  A[P[i].x + s].passid = i;
                  A[P[i].y+1+s].passid = -1;
                }

     				   P[i].res=0;
             }
             else
              P[i].res++;
            }
      			}
            else
                P[i].res=0;
      			//else dont move
     		}
     		else
     		{
     			if(dir==-1)
     			{
     				//move down
     				if(A[P[i].y+1].passid == -1 || A[P[i].y+2].passid == -1 || A[P[i].y+3].passid == -1 || A[P[i].y+4].passid == -1)
     				{
     					if(P[i].speed!=0.0f)
     					{
                if(P[i].res==2)
                {


                  P[i].x-=2*P[i].speed;
                  P[i].y-=2*P[i].speed;

                  for (int s = 0; s < 2*P[i].speed; ++s)
                  {
                    A[P[i].y - s].passid = i;
                    //A[P[i].x+1].passid = i;

                    A[P[i].x - 1 - s].passid = -1;
                    //A[P[i].y+2].passid = -1;
                  }
                  P[i].res=0;
     					  }
                else
                  P[i].res++;
              }
      			}
              else
                P[i].res=0;

     			}
     			else
     			{
            P[i].res=0;
     				//stay and jump to B or C
     				if(ex==0||ex==1||ex==6||ex==7)
     				{
     					//Going to B
              tex=ex;
     					if(ex==6||ex==7)
     						tex=ex-5;

              // Going to B[tex]

     					for(j=50;B[tex][j].passid==-1&&j> 50-P[i].diameter ;--j);

     					if(50 - j == P[i].diameter)
     					{
     						for(k=P[i].x;k<=P[i].y;++k)
                  A[k].passid=-1;
                P[i].x = tex;
     						P[i].y = j;

     					for(;j<=50;++j)
     						B[tex][j].passid=i;
              P[i].status=2;
     				 }
            }
     				else
     				{
              P[i].res=0;
                tex=ex-2;

              // Going to C[tex]

              for(j=50;C[tex][j].passid==-1&&j>50-P[i].diameter;--j);

              if(50 - j == P[i].diameter)
              {
                for(k=P[i].x;k<=P[i].y;++k)
                  A[k].passid=-1;
                P[i].x = tex;
                P[i].y = j;

              for(;j<=50;++j)
                C[tex][j].passid=i;
              P[i].status=3;
            }
     				}
     			}

     		}
     	break;


      case 2: // the Passenger is in midle of the exit front and end exit aisles i.e seat exit y represent the position in the aisle and
     		// x represent which aisle 1 / 2 / 3 / 4
     		if(P[i].y <= 0 && P[i].speed != 0.0f &&  (P[i].y-1 <= 0 || P[i].y-2 <= 0 || P[i].y-3 <= 0 || P[i].y-4 <= 0))
     		{
     	    for(j=0;j<10; ++j)
            {
              if(B[P[i].x][j].passid ==i)
                B[P[i].x][j].passid=-1;
          }
          P[i].status = 4; //Passenger is out of the plane
      	}
     		else
     		{

     			if(P[i].speed != 0.0f)
     			{
    				if(B[P[i].x][P[i].y-1].passid ==-1 || B[P[i].x][P[i].y-2].passid ==-1 || B[P[i].x][P[i].y-3].passid ==-1 || B[P[i].x][P[i].y-4].passid ==-1)
    				{
    	 				//move closer to the exit

    					P[i].y-=2*P[i].speed;

              for (int s = 0; s < 2*P[i].speed; ++s)
              {
                B[P[i].x][P[i].y + s].passid = i;
                B[P[i].x][(P[i].y + (int)P[i].diameter + 1 +s)].passid =-1;
              }
    	 			}
    	 		}
     		}

     	break;
     	case 3: // the Passenger is in middle of the middle exit aisles i.e seat exit y represent the position in the aisle and
     		// x represent which aisle 1 / 2 / 3 / 4
     	if(P[i].y <= 0 && P[i].speed != 0.0f && (P[i].y-1 <= 0 || P[i].y-2 <= 0 || P[i].y-3 <= 0 || P[i].y-4 <= 0))
     		{
     			for(j=0;j<10; ++j)
     				{
              if(C[P[i].x][j].passid ==i)
                C[P[i].x][j].passid=-1;
     			}
          P[i].status = 4; //Passenger is out of the plane
     		}
     		else
     		{

     			if(P[i].speed != 0.0f)
     			{
    				if(C[P[i].x][P[i].y-1].passid ==-1 || C[P[i].x][P[i].y-2].passid ==-1 || C[P[i].x][P[i].y-3].passid ==-1 || C[P[i].x][P[i].y-4].passid ==-1)
    				{
    	 				//move closer to the exit
    	 			 if(P[i].res==1)
             {
            	P[i].y-=2*P[i].speed;
              for (int s = 0; s < 2*P[i].speed; ++s)
              {
                C[P[i].x][P[i].y + s].passid = i;
                C[P[i].x][(P[i].y + (int)P[i].diameter + 1 +s)].passid =-1;
              }
            }
            else
              P[i].res++;
    	 			}
            else
              P[i].res=0;
    	 		}
     		}

     	break;
      case 4:
        P[i].x=-1;
        P[i].y=-1;
     };
    }
    }




void map_Passenger_to_exit(Passenger P[],int seat[][100], block C[][55],block B[][55],int h_exit[])
{
  int i,k,j,l;
  for(l=2;l<6;++l)
  {
    if(h_exit[l]==1)//the middle exit 2,3,4,5
    {
      if(l%2==0)
        {
          for(i=0;i<3;i++)
          {
            k=seat[15+(l-2)/2][i];
            if(k!=-1)
              {
                for(j=0;j<P[k].diameter;++j)
                  C[l-2][i*17 + j].passid=k;

                P[k].x=l-2;
                P[k].status = 3;
                P[k].y= i*17;
                P[k].res=0;
              }
            }
        }
        else
        {
          for(i=3;i<6;i++)
          {
            k=seat[15+(l-2)/2][i];
            if(k!=-1)
              {
                for(j=0;j<P[k].diameter;++j)
                  C[l-2][(5-i)*17 + j].passid=k;

                P[k].x=l-2;
                P[k].status = 3;
                P[k].y= (5-i)*17;
                P[k].res=0;
              }
            }
        }

    }
  }

}


//normal adjustment
int seatArrangement1(Passenger *h_P, int h_seat[][100], int numPass){
int r_row,r_col,i;
    for(i=0;i<numPass;++i)
    {
      r_row=rand()%30;   //should be defined in the header file TODO
      r_col=rand()%6;     //should be defined in the header file TODO
      if(h_seat[r_row][r_col]==-1)
        {
            h_P[i].x=r_row;
            h_P[i].y=r_col;
            if(r_row<0||r_col<0)
              {
                printf("Olala\n");
                return 0;
              }
            else
              h_seat[r_row][r_col]=i;
            printf("%d %d\n",r_row,r_col);
        }
      else
      {
          i--;
        }
    }
	return 1;
}

//older people in the aisle and younger in the window side
int seatArrangement2(Passenger *h_P, int h_seat[][100], int numPass){
  int r_row,r_col,i = 0,j =0;
      int oldPpl[100],youngPpl[100], k=0, l=0;
      for (i = 0; i < numPass;++i)
      {
        if (h_P[i].age > 40)
        {
          oldPpl[k] = i;
          ++k;
        }
        else{
          youngPpl[l] = i;
          ++l;
        }
      }
      int m =0, n=0;
      for (i = 0; i < 30; ++i)
      {
        for (j = 0; j < 6; ++j)
        {
          if (j%2 !=0 || j%3 == 0)
          {
            if(m<l)
            {
              h_P[youngPpl[m]].x = i;
              h_P[youngPpl[m]].y = j;
              h_seat[i][j] = youngPpl[m];
              ++m;
            }
          }
          else{
            if(n<k)
            {
              h_P[oldPpl[n]].x = i;
              h_P[oldPpl[n]].y = j;
              h_seat[i][j] = oldPpl[n];
              ++n;
            }
          }
        }
      }
    }
  return 1;
}

//younger people in the ailse and older in the window side
int seatArrangement3(Passenger *h_P, int h_seat[][100], int numPass){
int r_row,r_col,i = 0,j =0;
      int oldPpl[100],youngPpl[100], k=0, l = 0;
      for (i = 0; i < numPass;++i)
      {
        if (h_P[i].age > 40)
        {
          oldPpl[k] = i;
          ++k;
        }
        else{
          youngPpl[l] = i;
          ++l;
        }
      }
      int m = 0, n=0;
      for (i = 0; i < 30; ++i)
      {
        for (j = 0; j < 6; ++j)
        {
          if (j%2 !=0 || j%3 == 0)
          {
            if(m<k)
            {
              h_P[oldPpl[m]].x = i;
              h_P[oldPpl[m]].y = j;
              h_seat[i][j] = oldPpl[m];
              ++m;
            }
          }
          else{
            if(n<l)
            h_P[youngPpl[n]].x = i;
            h_P[youngPpl[n]].y = j;
            h_seat[i][j] = youngPpl[n];
            ++n;
          }
        }
      }
    }
  return 1;
}

// women in one half and men in second half
int seatArrangement4(Passenger *h_P, int h_seat[][100], int numPass){
  int r_row,r_col,i = 0,j =0;
      int women[100], men[100], k=0, l = 0;
      for (i = 0; i < numPass;++i)
      {
        if (h_P[i].sex == 0)
        {
          women[k] = i;
          ++k;
        }
        else{
          men[l] = i;
          ++l;
        }
      }

    for(i=0;i<k;++i)
    {
      r_row=rand()%15;
      r_col=rand()%6;
      if(h_seat[r_row][r_col]==-1)
        {
            h_P[women[i]].x=r_row;
            h_P[women[i]].y=r_col;
            if(r_row<0||r_col<0)
              {
                printf("Olala\n");
                return 0;
              }
            else
              h_seat[r_row][r_col]=i;
            //printf("%d %d\n",r_row,r_col);
        }
      else
      {
          i--;
        }
    }

    for(i=0;i<l;++i)
    {
      r_row=(rand()%15) + 15;
      r_col=rand()%6;
      if(h_seat[r_row][r_col]==-1)
        {
            h_P[men[i]].x=r_row;
            h_P[men[i]].y=r_col;
            if(r_row<0||r_col<0)
              {
                printf("Olala\n");
                return 0;
              }
            else
              h_seat[r_row][r_col]=i;
            //printf("%d %d\n",r_row,r_col);
        }
      else
      {
          i--;
        }
    }
return 1;
}

  //Main
  int main()
  {
    srand(time((0)));
    Aircraft air;
    char name[20];	
    printf("enter the name of the aircraft : ");
	scanf("%s",&name);
    air = aircraftInput(name);
   // printf("details aircraft:::row row %d ",air.row);
    int numPass,i,j;
    int count1=0;
printf("\n enter the number of passengers : ");
    scanf("%d",&numPass);
    Passenger *h_P =(Passenger *)malloc(sizeof(Passenger)*numPass);
    Passenger *P;


    pass_input(h_P,numPass);

  	// Seating Arrangement Assigning Each Passenger location to sit randomely
  	// Think something to make sure the random function does not send it to infinite loop

  	int h_seat[100][100];
  	int (*seat)[100];

  	for(i=0;i<100;++i)
  	{
  		for(j=0;j<100;++j)
  			h_seat[i][j]=-1;
  	}

  	//all seats are vacant right now
	int tt = seatArrangement1(h_P, h_seat, numPass);
  	if(tt == 0){
		return 0;
	}

  /*
    for(i=0;i<30;++i)
    {
      for(j=0;j<10;++j)
        printf("%d ",h_seat[i][j]+1);
      printf("\n");
    }
*/


  	// Now each row is occupied by some Passengers
  	// Each Passenger is sitting in a row and each row is having corresponding aisle array portion in front of it.
  	// The Passenger can move to the aisle A[] in front of its row if it is unoccupied
  printf("Seating Done\n");

  	block h_A[1040];
  	block* A;


  	// A is the aisle
  	//Each Element of the
  	for(j=0;j<1040;++j)
  		{
  				h_A[j].passid=-1;
  				h_A[j].exit=0;
  		}
  	// the aisle is empty right now
  	//Now there are 4 Normal Gate Exits and
  	//Exit Paths are of 2 types 1 end and other in the middle each one will have different speeds


  	block h_B[4][55]; // Nornal Exit Paths 2 on each ends of the plane
  	block h_C[4][55]; // Seat Exit 2 in the middle of the plane

  	block (*B)[55]; // Nornal Exit Paths 2 on each ends of the plane
  	block (*C)[55]; // Seat Exit 2 in the middle of the plane

  	for(i=0;i<4;++i)
  	{
  		for(j=0;j<55;++j)
  			{
  				h_B[i][j].passid=-1;
  				h_C[i][j].passid=-1;
  				h_B[i][j].exit=0;
  				h_C[i][j].exit=0;
  			}
  	}

  	// set up the exits all the B exits are empty
  	// C exits or the middle exits are occupied by people
  	//int h_exit[6] = {1,1,1,1,1,1};
  	//exit is 1 for those wxits which are open and 0 for those which are close
  	//block A[],block B[4][55],block C[4][55],Passenger P[] ,int seat[][100],int numPass
  	//hipMalloc((void **) &array1_d , WIDTH*WIDTH*sizeof (int) ) ;
  	// Here the game starts
  	//Emergency! Emergency! Emergency! Run all of you Out of the plane
    printf("Enter 1 if the exit is open and 0 if the exit is close for all the 8 exits");
    int h_exit[8];
    int *d_exit;
    for(i=0;i<8;++i)
      scanf("%d",&h_exit[i]);

  	//__global__ void movement_to_exit(block A[],block B[4][55],block C[4][55],Passenger P[] ,int seat[][100],int numPass) //runs for each Passenger and make his movmenent according to the positions
  	int numout=0,numprev=0;
  	j=0;
    hipMalloc((void **) &P , numPass*sizeof (Passenger) ) ;
    hipMalloc((void **) &B , (55*4)*sizeof (block) ) ;
    hipMalloc((void **) &C , (55*4)*sizeof (block) ) ;
    hipMalloc((void **) &A , 1040*sizeof (block) ) ;
    hipMalloc((void **) &seat , (100*100)*sizeof (int) ) ;
    hipMalloc((void **) &d_exit , (8)*sizeof (int) ) ;

    // select exit
    map_Passenger_to_exit(h_P,h_seat, h_C,h_B,h_exit);
   /* for(j=0;j<4;++j)
    {
      for(i=0;i<55;++i)
      {
       printf("%d",h_C[j][i]);
      }
      printf("\n");
    }*/
    count1=0;
int filecounter = 0;  	
while(numout<numPass)
  	 {
++filecounter;
     // ifiiii(j==1000)
      //  break;
      numprev=numout;
  		numout=0;

   /* for(i=0;i<30;++i)
    {
      for(j=0;j<10;++j)
        printf("%d ",h_seat[i][j]+1);
      printf("\n");
    }
    */

/*    for(i=0;i<numPass;++i)
      {
        if(P[i].status!=4)
        printf("Passengr %d : (%d,%d) : %d : ans : %d : dir : %d\n",i,h_P[i].x,h_P[i].y,h_P[i].status,h_P[i].ans,h_P[i].dir);
      }

  */
    //printf("Passengr %d : (%d,%d) : %d : ans : %d : dir : %d\n",3,h_P[3].x,h_P[3].y,h_P[3].status,h_P[3].ans,h_P[3].dir);

    hipMemcpy ( P , h_P , numPass*sizeof (Passenger) , hipMemcpyHostToDevice);
    hipMemcpy ( seat , h_seat , 100*100*sizeof (int) , hipMemcpyHostToDevice);
    hipMemcpy ( C , h_C , 4*55*sizeof (block) , hipMemcpyHostToDevice);
    hipMemcpy ( B , h_B , 4*55*sizeof (block) , hipMemcpyHostToDevice);
    hipMemcpy ( A , h_A , 1040*sizeof (block) , hipMemcpyHostToDevice);
    hipMemcpy ( d_exit , h_exit , 8*sizeof (int) , hipMemcpyHostToDevice);


      movement_to_exit<<< 1,numPass >>>(A,B,C,P,seat,d_exit,numPass);

  		hipError_t err1 = hipPeekAtLastError();
      hipDeviceSynchronize();
      //printf( "Got CUDA error ... %s \n", hipGetErrorString(err1));

    hipMemcpy ( h_P , P , numPass*sizeof (Passenger) , hipMemcpyDeviceToHost);
    hipMemcpy ( h_seat , seat , 100*100*sizeof (int) , hipMemcpyDeviceToHost);
    hipMemcpy ( h_C , C , 4*55*sizeof (block) , hipMemcpyDeviceToHost);
    hipMemcpy ( h_B , B , 4*55*sizeof (block) , hipMemcpyDeviceToHost);
    hipMemcpy ( h_A , A , 1040*sizeof (block) , hipMemcpyDeviceToHost);


// creating file
  // FILE *fp = NULL;

    char filename[50] = "res";
    char str[20];
    sprintf(str, "%03d", filecounter);
//	printf("filecounter checkup %s",str);    
	strcat(filename,str);
    strcat(filename,".res");
float tt = (j*40.6) + 7000.0;
createGlobalMatrix(h_seat, h_A, h_B, h_C, filename,tt);
/*
FILE *fp = NULL;
    fp = fopen(filename, "w");
        fprintf(fp," -----------------------------\n"); 
        fprintf(fp, "\n");
	fprintf(fp, "%f", j*40);
        fprintf(fp,"\n ----------------------------- \n"); 
        fprintf(fp,"passenger \n"); 
        fprintf(fp,"\t \t 1000 \t  1 \n"); 
	for (int d = 0; d < 1000; ++d)
      {
	if(h_A[d].passid == -1)
	{
		fprintf(fp, "0");
       		fprintf(fp, "\n");
	}
	else{
		fprintf(fp,"1");
		fprintf(fp, "\n");
        }
}
fprintf(fp,"\n ----------------------------- \n");
    fclose(fp);
*/

    //printToFile(filename,h_A);
//file creation complete

  		for(i=0;i<numPass;i++)
  		{
  			if(h_P[i].status == 4)
  				numout++;
  		 //printf("%d\n",h_P[i].status);
      }

    if(numprev==numout)
      {
        printf("*");
        count1++;
      }
    else
    {
      count1=0;
      printf("%d %d\n",numout,j);
    }
      if(numout==numPass)
        break;

    //printf("%d %d\n",numout,j);

     /* if(count1>100)
      {
        for(i=0;i<numPass;i++)
        {
          if(h_P[i].status!=4)
          {
                printf("Passenger %d : (%d,%d) : %d : ans : %d : dir : %d\n",i,h_P[i].x,h_P[i].y,h_P[i].status,h_P[i].ans,h_P[i].dir);
          }
        }
        break;
      }
*/
     // printf("%d\t %d\n",numout,j);
  		j++;
    
   // printf("jjjj : %d\n", j);
  	}

  	float timeSteps = 40.6;
	printf("%f\n",j);
  	printf("total time:  %f\n",j*timeSteps + 7000.0);
  //  printf("%f\n",j*timeSteps);
   // printf("timeSteps : %f\n",j);
    printf("filecounter : %d\n", filecounter);
  	return 0;
  }


  int random(int min,int max);

  void pass_input(Passenger *P,int n)
  {
    int i;//r;
    srand(time(0));
    Passenger *tp=P;

    for(i=0;i<n;++i,tp++)
      {
        //r=rand();
        //  printf("%d\n",r);
        tp->id=i;
        //  tp->x=
        //
        tp->sex=random(0,1); // Male or female(random 0-1)
        tp->status = 0;
        tp->Mtime=tp->sex?random(1000,1750):random(1100,1950);
        tp->Wtime=50;
        tp->Rtime=random(400,700); // Random  (500-1000)ms
        tp->fear=-1; //fear value 0
        tp->agility=-1; // agility value

        tp->diameter=7;  //random(8,10);//(Random ) // diameter occupied by passenger
        tp->totaltime=0; //total time to evacuate
        tp->totalDist=0; //total distance to exit
        
	if(tp->sex==0)
         tp->speed=1.5f;  //Random (1-1.5 ) speed of passenger
        else
          tp->speed=1.0f;

        tp->grpstatus=-1; // Not in this paper in group or not
        tp->timeSteps=178; // minimum unit of time = 178 miliseconds
        tp->res=0;
        printf("id : %d , sex : %d , Mtime : %d, Rtime : %d\n",tp->id,tp->sex,tp->Mtime,tp->Rtime);
      }

  }


Aircraft aircraftInput(char name[]){
Aircraft A;   
 char filename1[100];
strcpy(filename1,name);    
strcat(filename1,".dat");

   FILE *fp;
   char buff[255];
   int res[10];
   int k =0;
   fp = fopen(filename1, "r");
   while(fgets(buff, 80, fp) != NULL)
   {
   //fscanf(fp, "%s", buff);
   //printf("%s\n", buff );
   int result = atoi(buff);
   //printf("%d\n", result);
    res[k] = result;
    ++k;

    }
    fclose(fp);
   /*for(int u =0;u<4;++u){
	printf(" \n %d",res[u]);  
 }*/
   A.row = res[0];
   A.column = res[1];
   A.numOfExitPassage = res[2];
   A.maxNumPassenger = res[3];
   return A;
}

void createGlobalMatrix(int h_seat[100][100], block h_A[1040], block h_B[4][55], block h_C[4][55], char filename[50],float tt){
int global[1040][130];
//printf("\n 1. we reached here\n");
//printf("%s","testing");
    //main exit 1
    int k =0, i=0;
//printf("\n1.  i= %d",i);
    for(k =0;k<50;++k){
      for (i = 0; i < 55; ++i)
      {
          if(h_B[0][i].passid==-1){
            global[k][i] = 0;
          }
          else{
            global[k][i] = 1;
          }
        }
       // int temp = i;
        if(h_A[k].passid == -1){
            global[k][i] = 0;
          }
          else{
            global[k][i] = 1;
          }
	int temp = i;
      for (i = 54; i >=0; --i)
      {
          if(h_B[1][i].passid == -1){
            global[k][temp] = 0;
          }
          else{
            global[k][temp] = 1;
          }
        ++temp;
	}
      }
//printf("\n2. i= %d",i);
      // 1-14 seats
//printf("\n 3. we reached here\n");      
int p = 0;
      for (i = 50; i < (14*30)+50; ++i)
      {
          int u=0;
        for (int j = 0; j < 3; ++j)
        {
          if (h_seat[p][j] == -1)
          {
            for (; u < 18*(j+1); ++u)
            {
              global[i][u] = 0;
            }
          }
          else{
              for (; u < 18*(j+1); ++u)
              {
                global[i][u] = 1;
              }
          }
        }
        
	for(int po =0;po<18;++po){
		if(h_A[i].passid == -1)
			global[i][u] = 0;
        	else
			global[i][u] = 1;
		++u;
	}
        u=0;
        for (int j = 3; j < 6; ++j)
        {

          if (h_seat[p][j] == -1)
          {
            for (; u < 18*(j+1); ++u)
            {
              global[i][u] = 0;
            }
          }
          else{
              for (; u < 18*(j+1); ++u)
              {
                global[i][u] = 1;
              }
          }
        }
        if ((i+1-50)%30 == 0)
        {
          ++p;
        }
      }
//printf("\n3. i= %d",i);

      //middle exits
//printf("\n 4. we reached here\n");
      for(k =i;k<i+50;++k){
      int j =0;
      for (j = 0; j < 55; ++j)
      {
          if(h_C[0][j].passid == -1){
            global[k][j] = 0;
          }
          else{
            global[k][j] = 1;
          }
        }
	for(int po =0;po<18;++po){
                if(h_A[i].passid == -1)
                        global[i][j] = 0;
                else
                        global[i][j] = 1;
                ++j;
        }
        int temp = j;
        

      for (j = 54; j >=0; --j)
      {
          if(h_C[1][j].passid == -1){
            global[k][temp] = 0;
          }
          else{
            global[k][temp] = 1;
          }
         ++temp;
	}
      }

      // middle exits  correct the second loop for j.. j is not in the sync
//printf("\n4. i= %d k = %d",i,k);
      for(;k<i+100;++k){
      int j=0;
      for (j = 0; j < 55; ++j)
      {
          if(h_C[2][j].passid == -1){
            global[k][j] = 0;
          }
          else{
            global[k][j] = 1;
          }
        }
        
        for(int po =0;po<18;++po){
                if(h_A[i].passid == -1)
                        global[i][j] = 0;
                else
                        global[i][j] = 1;
                ++j;
        }
        int temp = j;

      for (j = 54; j >=0; --j)
      {
          if(h_C[3][j].passid == -1){
            global[k][temp] = 0;
          }
          else{
            global[k][temp] = 1;
          }
        }
      }

//printf("\n 5. we reached here\n");
      // 17 - 30 seats
//printf("\n5. i= %d",i);
      for (i = k; i < (28*30)+150; ++i)
      {
          int u=0;
        for (int j = 0; j < 3; ++j)
        {
          if (h_seat[p][j] == -1)
          {
            for (; u < 18*(j+1); ++u)
            {
              global[i][u] = 0;
            }
          }
          else{
              for (; u < 18*(j+1); ++u)
              {
                global[i][u] = 1;
              }
          }
        }
        for(int po =0;po<18;++po){
                if(h_A[i].passid == -1)
                        global[i][u] = 0;
                else
                        global[i][u] = 1;
                ++u;
        }
        //int temp = u;
        //u=0;
        for (int j = 3; j < 6; ++j)
        {

          if (h_seat[p][j] == -1)
//        fprintf(fp, "\n");
          {
            for (; u < 18*(j+2); ++u)
            {
              global[i][u] = 0;
            }
          }
          else{
              for (; u < 18*(j+2); ++u)
              {
                global[i][u] = 1;
              }
          }
        }
        if ((i+1-150)%30 == 0)
        {
          ++p;
        }
      }
//printf("%d",i);
//printf("\n 6. we reached here\n");

//end exits

//printf("%d",i);
k =i;      
//int q = k+50;
     
	for(int ty = 0;ty<50;++ty){
     // int i =0;
      for (i = 0; i < 55; ++i)
      {
//	printf("\n1111");
//printf("value of %d",i);	          
	if(h_B[2][i].passid == -1){
            global[k][i] = 0;
          }
          else{
            global[k][i] = 1;
          }
        }
//	printf("\n2");

        
	for(int po =0;po<18;++po){
                if(h_A[k].passid == -1)
                        global[k][i] = 0;
                else
                        global[k][i] = 1;
                ++i;
        }
        int temp = i;


//	printf("\n3");

      for (i = 54; i >=0; --i)
      {
  //      printf("\n4");
  
	if(h_B[3][i].passid == -1){
            global[k][temp] = 0;
          }
          else{
            global[k][temp] = 1;
          }
	++temp;
        }
++k;
//	printf("\n5");
//	printf("\n k = %d",k);
      }
//printf("\n 2. we reached here\n");
FILE *fp = NULL;
    fp = fopen(filename, "w");
        //fprintf(fp," -----------------------------\n");
        //fprintf(fp, "\n");
        //fprintf(fp, "   %f", tt);
        //fprintf(fp,"\n -----------------------------\n");
        //fprintf(fp," passenger evacuation\n");
        //fprintf(fp,"         130          1040\n");
        for (int d = 0; d < 1040; ++d)
        {
        for(int g = 0; g < 130; ++g){
                fprintf(fp, "%d,", global[d][g]);
	}
	fprintf(fp,"\n");
	}
//fprintf(fp," -----------------------------\n");
    fclose(fp);

}

  int random(int min,int max)
  {
    int r = rand();
    r=r%(max-min+1);
    r=r+min;
    return r;
  }
  void printToFile(char filename[], int a[]){

  FILE *fp = NULL;
    fp = fopen(filename, "w");
      for (int j = 0; j < 1040; ++j)
      {
        fprintf(fp, "%d", a[j]);
	fprintf(fp, "\n");      
	}

    fclose(fp);

  }

