#include "hip/hip_runtime.h"
#include<stdio.h>
#include"Passenger.h"
#include"Aircraft.h"

int random(int min,int max);

void pass_input(Passenger *P,int n)
{
  int i;//r;
  srand(time(0));
  Passenger *tp=P;
  
  for(i=0;i<n;++i,tp++)
    {
      //r=rand();
      //  printf("%d\n",r);
      tp->id=i;
      //  tp->x=
      //
      tp->sex=random(0,1); // Male or female(random 0-1)
      tp->status = 0;
      tp->Mtime=tp->sex?random(875,1750):random(920,1950);
      tp->Wtime=50;
      tp->Rtime=random(400,700); // Random  (500-1000)ms
      tp->fear=-1; //fear value 0
      tp->agility=-1; // agility value
      tp->diameter=-1;//(Random ) // diameter occupied by passenger
      tp->totaltime=0; //total time to evacuate
      tp->totalDist=0; //total distance to exit
      tp->speed=0;  //Random (1-1.5 ) speed of passenger
      tp->grpstatus=-1; // Not in this paper in group or not
      tp->timeSteps=178; // minimum unit of time = 178 miliseconds
      printf("id : %d , sex : %d , Mtime : %d, Rtime : %d\n",tp->id,tp->sex,tp->Mtime,tp->Rtime);
    }
  
}
int random(int min,int max)
{
  int r = rand();
  r=r%(max-min+1);
  r=r+min;
  return r;
}
